#include "hip/hip_runtime.h"
#include "MultiplyGPU.h"

namespace OFX {
namespace Plugin {
namespace MultiplyGPU {

__global__ void CUDAKernel(int p_Width, int p_Height, float p_ValueR, float p_ValueG, float p_ValueB, float p_ValueA, const float* p_Input, float* p_Output)
{
   const int x = blockIdx.x * blockDim.x + threadIdx.x;
   const int y = blockIdx.y * blockDim.y + threadIdx.y;

   if ((x < p_Width) && (y < p_Height))
   {
       const int index = ((y * p_Width) + x) * 4;

       p_Output[index + 0] = p_Input[index + 0] * p_ValueR;
       p_Output[index + 1] = p_Input[index + 1] * p_ValueG;
       p_Output[index + 2] = p_Input[index + 2] * p_ValueB;
       p_Output[index + 3] = p_Input[index + 3] * p_ValueA;
   }
}

void RunCUDAKernel(int p_Width, int p_Height, const float* p_Value, const float* p_Input, float* p_Output)
{
    dim3 threads(128, 1, 1);
    dim3 blocks(((p_Width + threads.x - 1) / threads.x), p_Height, 1);

    CUDAKernel<<<blocks, threads>>>(p_Width, p_Height, p_Value[0], p_Value[1], p_Value[2], p_Value[3], p_Input, p_Output);
}

} // namespace MultiplyGPU {
} // namespace Plugin {
} // namespace OFX {
