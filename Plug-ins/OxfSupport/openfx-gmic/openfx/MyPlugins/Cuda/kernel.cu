
#include <hip/hip_runtime.h>
__global__ void gain(int width, int height, float rGain, float gGain, float bGain, float* input, float* output)
{
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;

   if ((x < width) && (y < height))
   {
       int index = (y * width + x) * 4;
       output[index + 0] = input[index + 0] * rGain;
       output[index + 1] = input[index + 1] * gGain;
       output[index + 2] = input[index + 2] * bGain;
       output[index + 3] = input[index + 3];
   }
}

void RunKernel(int p_Width, int p_Height, float p_RGain, float p_GGain, float p_BGain, float* p_Input, float* p_Output)
{
    dim3 threads(128, 1, 1);
    dim3 blocks((((p_Width + threads.x - 1) / threads.x) * threads.x), p_Height, 1);

    gain<<<blocks, threads>>>(p_Width, p_Height, p_RGain, p_GGain, p_BGain, p_Input, p_Output);
}
